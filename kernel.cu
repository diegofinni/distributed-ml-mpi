#include "hip/hip_runtime.h"
__global__ void dot_CUDA(int *a, int *b, int *c) {
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    temp[threadIdx.x] = a[index] * b[index];

    __syncthreads();

    if( 0 == threadIdx.x ) {
        int sum = 0;
        for( int i = 0; i < THREADS_PER_BLOCK; i++ )
            sum += temp[i];
        atomicAdd(xc , sum );
    }
}